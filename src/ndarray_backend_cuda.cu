#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>
#include <unordered_map>
#include <vector>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

std::unordered_map<size_t, std::vector<scalar_t*>> buffer;

struct CudaArray {
  CudaArray(const size_t size) {
    if(buffer[size].size()){
      ptr = buffer[size].back();
      buffer[size].pop_back();
    }
    else {

      hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
      if (err != hipSuccess){
        throw std::runtime_error(hipGetErrorString(err));
      }
    }
    this->size = size;
  }
  ~CudaArray() { 
    hipFree(ptr);
    // buffer[size].push_back(ptr);
  }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  int32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<int32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides



__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN SOLUTION
  if(gid < size){
    size_t t = gid;
    size_t sz = shape.size - 1;
    while(t){
      offset += (t % shape.data[sz]) * strides.data[sz];
      t /= shape.data[sz];
      sz--;
    }
    out[gid] = a[offset];
  }
  /// END SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
             std::vector<int32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}


__global__ void EwiseSetitemKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                                   CudaVec strides, size_t offset){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size){
    size_t t = gid;
    size_t sz = shape.size - 1;
    while(t){
      offset += (t % shape.data[sz]) * strides.data[sz];
      t /= shape.data[sz];
      sz--;
    }
    out[offset] = a[gid];
  }
}

void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
                  std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  EwiseSetitemKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, a.size, VecToCuda(shape),
                                              VecToCuda(strides), offset);
  /// END SOLUTION
}


__global__ void ScalarSetitemKernel(scalar_t a, scalar_t* out, size_t size, CudaVec shape,
                                   CudaVec strides, size_t offset){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size){
    size_t t = gid;
    size_t sz = shape.size - 1;
    while(t){
      offset += (t % shape.data[sz]) * strides.data[sz];
      t /= shape.data[sz];
      sz--;
    }
    out[offset] = a;
  }
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<int32_t> shape,
                   std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ScalarSetitemKernel<<<dim.grid, dim.block>>>(val, out->ptr, size, VecToCuda(shape),
                                              VecToCuda(strides), offset);
  /// END SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

// __global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
//   size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
//   if (gid < size) out[gid] = a[gid] + b[gid];
// }

// void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
//   /**
//    * Add together two CUDA array
//    */
//   CudaDims dim = CudaOneDim(out->size);
//   EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
// }

// __global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
//   size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
//   if (gid < size) out[gid] = a[gid] + val;
// }

// void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
//   /**
//    * Add together a CUDA array and a scalar value.
//    */
//   CudaDims dim = CudaOneDim(out->size);
//   ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
// }

#define def_ewise_binop_cuda_kernel(name, op) \
__global__ void Ewise##name##Kernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) { \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = a[gid] op b[gid]; \
}

#define def_ewise_binop_cuda(name) \
void Ewise##name(const CudaArray& a, const CudaArray& b, CudaArray* out) { \
  CudaDims dim = CudaOneDim(out->size); \
  Ewise##name##Kernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size); \
}

#define def_scalar_binop_cuda_kernel(name, op) \
__global__ void Scalar##name##Kernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) { \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = a[gid] op val; \
}

#define def_scalar_binop_cuda(name) \
void Scalar##name(const CudaArray& a, scalar_t val, CudaArray* out) { \
  CudaDims dim = CudaOneDim(out->size); \
  Scalar##name##Kernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size); \
}

#define def_ewise_binfunc_cuda_kernel(name, binfunc) \
__global__ void Ewise##name##Kernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size){ \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = binfunc(a[gid], b[gid]); \
}

#define def_ewise_binfunc_cuda(name) \
void Ewise##name(const CudaArray& a, const CudaArray& b, CudaArray* out) { \
  CudaDims dim = CudaOneDim(out->size); \
  Ewise##name##Kernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size); \
}

#define def_scalar_binfunc_cuda_kernel(name, binfunc) \
__global__ void Scalar##name##Kernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size){ \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = binfunc(a[gid], val); \
}

#define def_scalar_binfunc_cuda(name) \
void Scalar##name(const CudaArray& a, scalar_t val, CudaArray* out) { \
  CudaDims dim = CudaOneDim(out->size); \
  Scalar##name##Kernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size); \
}

#define def_ewise_ufunc_cuda_kernel(name, ufunc) \
__global__ void Ewise##name##Kernel(const scalar_t* a, scalar_t* out, size_t size){ \
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x; \
  if (gid < size) out[gid] = ufunc(a[gid]); \
}

#define def_ewise_ufunc_cuda(name) \
void Ewise##name(const CudaArray& a, CudaArray* out) { \
  CudaDims dim = CudaOneDim(out->size); \
  Ewise##name##Kernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size); \
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */


////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

/* element wise binary operations */
def_ewise_binop_cuda_kernel(Add, +)
def_ewise_binop_cuda(Add)

def_ewise_binop_cuda_kernel(Mul, *)
def_ewise_binop_cuda(Mul)

def_ewise_binop_cuda_kernel(Div, /)
def_ewise_binop_cuda(Div)

def_ewise_binop_cuda_kernel(Eq, ==)
def_ewise_binop_cuda(Eq)

def_ewise_binop_cuda_kernel(Ge, >=)
def_ewise_binop_cuda(Ge)

/* element wise binary operations */
def_scalar_binop_cuda_kernel(Add, +)
def_scalar_binop_cuda(Add)

def_scalar_binop_cuda_kernel(Mul, *)
def_scalar_binop_cuda(Mul)

def_scalar_binop_cuda_kernel(Div, /)
def_scalar_binop_cuda(Div)

def_scalar_binop_cuda_kernel(Eq, ==)
def_scalar_binop_cuda(Eq)

def_scalar_binop_cuda_kernel(Ge, >=)
def_scalar_binop_cuda(Ge)

/* element wise call binary functions */
def_ewise_binfunc_cuda_kernel(Maximum, max)
def_ewise_binfunc_cuda(Maximum)

/* call binary functions with scalar */
def_scalar_binfunc_cuda_kernel(Maximum, max)
def_scalar_binfunc_cuda(Maximum)

def_scalar_binfunc_cuda_kernel(Power, powf)
def_scalar_binfunc_cuda(Power)

/* element wise call unary functions */
def_ewise_ufunc_cuda_kernel(Log, logf)
def_ewise_ufunc_cuda(Log)

def_ewise_ufunc_cuda_kernel(Exp, expf)
def_ewise_ufunc_cuda(Exp)

def_ewise_ufunc_cuda_kernel(Tanh, tanhf)
def_ewise_ufunc_cuda(Tanh)


// #define min(a, b) ((a) < (b) ? (a) : (b))

/*
  Naive version of matmul
*/

__global__ void MatmulKernel_00(const scalar_t* a, const scalar_t* b, scalar_t* out, uint32_t M, uint32_t N, uint32_t P){
  int tx = blockIdx.x * blockDim.x + threadIdx.x;
  int ty = blockIdx.y * blockDim.y + threadIdx.y;
  if(ty < M && tx < P) {
      scalar_t c = 0;
      for(int i = 0; i < N; ++i){
          c += a[ty * N + i] * b[i * P + tx];
      }
      out[ty * P + tx] = c;
  }
}

/*
  simple using of shared memory
*/
#define KERNEL1_BLOCK_SIZE 16
__global__ void MatmulKernel_01(const scalar_t* a, const scalar_t* b, scalar_t* out, uint32_t M, uint32_t N, uint32_t P){
  const int blockRow = blockIdx.y;
  const int blockCol = blockIdx.x;

  // which block
  scalar_t* outSub = &out[(blockRow * P + blockCol) * KERNEL1_BLOCK_SIZE];
  size_t outDimX = min(KERNEL1_BLOCK_SIZE, P - blockCol * KERNEL1_BLOCK_SIZE);
  size_t outDimY = min(KERNEL1_BLOCK_SIZE, M - blockRow * KERNEL1_BLOCK_SIZE);

  __shared__ scalar_t as[KERNEL1_BLOCK_SIZE][KERNEL1_BLOCK_SIZE];
  __shared__ scalar_t bs[KERNEL1_BLOCK_SIZE][KERNEL1_BLOCK_SIZE];

  scalar_t val = 0.;
  for(size_t k = 0; k < N; k += KERNEL1_BLOCK_SIZE){
    __syncthreads();
    size_t innerDim = min((int)KERNEL1_BLOCK_SIZE, (int)(N - k));
    const scalar_t* aSub = &a[blockRow * KERNEL1_BLOCK_SIZE * N + k];
    const scalar_t* bSub = &b[k * P + blockCol * KERNEL1_BLOCK_SIZE];
    
    if(threadIdx.x < innerDim && threadIdx.y < outDimY){
      as[threadIdx.y][threadIdx.x] = aSub[threadIdx.y * N + threadIdx.x];
    }
    if(threadIdx.x < outDimX && threadIdx.y < innerDim){
      bs[threadIdx.y][threadIdx.x] = bSub[threadIdx.y * P + threadIdx.x];
    }
    __syncthreads();

    #pragma unroll
    for(size_t e = 0; e < innerDim; ++e){
      if(threadIdx.y < outDimY && threadIdx.x < outDimX){
        val += as[threadIdx.y][e] * bs[e][threadIdx.x];
      }
    }
  }
  if(threadIdx.y < outDimY && threadIdx.x < outDimX)
    outSub[threadIdx.y * P + threadIdx.x] = val;
}

/*
  each thread computes a tile * tile submatrix
*/
#define T 8
#define BLOCK_SIZE 128
#define INNER_STEP 8
__global__ void MatmulKernel_02(const scalar_t* a, const scalar_t* b, scalar_t* out, int M, int N, int P){
  int bx = blockIdx.x;
  int by = blockIdx.y;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int outBlockBaseOff = (by * P + bx) * BLOCK_SIZE;
  int outBlockDimX = min(BLOCK_SIZE, P - bx * BLOCK_SIZE);
  int outBlockDimY = min(BLOCK_SIZE, M - by * BLOCK_SIZE);

  // using int for threads whose dim is negtive
  int tileDimX = min(T, outBlockDimX - tx * T);
  int tileDimY = min(T, outBlockDimY - ty * T);

  scalar_t c[T][T] = {0.};
  __shared__ scalar_t as[INNER_STEP][BLOCK_SIZE];
  __shared__ scalar_t bs[INNER_STEP][BLOCK_SIZE];


  int tid = ty * blockDim.x + tx;
  int a_i = tid >> 1;
  int a_j = (tid & 1) << 2;
  int b_i = tid >> 5;
  int b_j = (tid & 31) << 2;
  for(int k = 0; k < N; k += INNER_STEP){
    int aBlockBaseOff = by * BLOCK_SIZE * N + k;
    int bBlockBaseOff = k * P + bx * BLOCK_SIZE;
    int innerDim = min(INNER_STEP, N - k);

    if(a_i < outBlockDimY && a_j < innerDim){
      if((N & 3) || innerDim - a_j < 4){
        // not aligned to 128 bytes
        // or no 4 more elements
        for(int t = 0; t < min(innerDim - a_j, 4); ++t){
          as[a_j + t][a_i] = a[aBlockBaseOff + a_i * N + a_j + t];
        }
      }
      else{
        float4 load_a_reg = reinterpret_cast<const float4*>(&a[aBlockBaseOff + a_i * N + a_j])[0];
        as[a_j][a_i] = load_a_reg.x;
        as[a_j + 1][a_i] = load_a_reg.y;
        as[a_j + 2][a_i] = load_a_reg.z;
        as[a_j + 3][a_i] = load_a_reg.w;
      }
    }
    
    if(b_i < innerDim && b_j < outBlockDimX){
      if((P & 3) || outBlockDimX - b_j < 4){
        // not aligned to 128 bytes
        // or no 4 more elements
        for(int t = 0; t < min(outBlockDimX - b_j, 4); ++t){
          bs[b_i][b_j + t] = b[bBlockBaseOff + b_i * P + b_j + t];
        }
      }
      else{
        reinterpret_cast<float4*>(&bs[b_i][b_j])[0] = reinterpret_cast<const float4*>(&b[bBlockBaseOff + b_i * P + b_j])[0];
      }
    }
    
    // sync for loading data into shared memory
    __syncthreads();
    float compute_a_reg[T];
    float compute_b_reg[T];

    for(int e = 0; e < innerDim; ++e){
        reinterpret_cast<float4*>(&compute_a_reg[0])[0] = reinterpret_cast<float4*>(&as[e][ty * T])[0];
        reinterpret_cast<float4*>(&compute_a_reg[4])[0] = reinterpret_cast<float4*>(&as[e][ty * T + 4])[0];
        reinterpret_cast<float4*>(&compute_b_reg[0])[0] = reinterpret_cast<float4*>(&bs[e][tx * T])[0];
        reinterpret_cast<float4*>(&compute_b_reg[4])[0] = reinterpret_cast<float4*>(&bs[e][tx * T + 4])[0];
        for(int i = 0; i < tileDimY; ++i){
            for(int j = 0; j < tileDimX; ++j){
                c[i][j] += compute_a_reg[i] * compute_b_reg[j];
            }
        }
    }
    __syncthreads();
  }

  int outTileBaseOff = outBlockBaseOff + ty * T * P + tx * T;
  if(P & 3) {
    for(int i = 0; i < tileDimY; ++i){
      for(int j = 0; j < tileDimX; ++j){
        out[outTileBaseOff + i * P + j] = c[i][j];
      }
    }
  }
  else {
    for(int i = 0; i < tileDimY; ++i){
      for(int j = 0; j < tileDimX; j+=4){
        reinterpret_cast<float4*>(&out[outTileBaseOff + i * P + j])[0] = reinterpret_cast<const float4*>(&c[i][j])[0];
      }
    }
  }
}


void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN SOLUTION
  // int kernel = 2;
  // switch(kernel){
  //   case 0: {
  //     // invoking kernel_00
  //     dim3 dimGrid((P + 15) / 16, (M + 15) / 16, 1);
  //     dim3 dimBlock(16, 16, 1);
  //     MatmulKernel_00<<<dimGrid, dimBlock>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  //     break;
  //   }
  //   case 1: {
  //     // invoking kernel_01
  //     size_t BM = (M + KERNEL1_BLOCK_SIZE - 1) / KERNEL1_BLOCK_SIZE;
  //     size_t BP = (P + KERNEL1_BLOCK_SIZE - 1) / KERNEL1_BLOCK_SIZE;
  //     dim3 grid = dim3(BP, BM, 1);
  //     dim3 block = dim3(KERNEL1_BLOCK_SIZE, KERNEL1_BLOCK_SIZE, 1);
  //     MatmulKernel_01<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  //     break;
  //   }
  //   case 2: {
  //     // invoking kernel_02
  //     size_t BM = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
  //     size_t BP = (P + BLOCK_SIZE - 1) / BLOCK_SIZE;
  //     dim3 grid = dim3(BP, BM);
  //     dim3 block = dim3(BLOCK_SIZE / T, BLOCK_SIZE / T);
  //     MatmulKernel_02<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  //     break;
  //   }
  //   default: assert(0);
  // }

  // if(M % 8 == 0 && N % 8 == 0 && P % 8 == 0 && 
  //     M >= 512 && M == N && P == N){
  //   size_t BM = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
  //   size_t BP = (P + BLOCK_SIZE - 1) / BLOCK_SIZE;
  //   dim3 grid = dim3(BP, BM);
  //   dim3 block = dim3(BLOCK_SIZE / T, BLOCK_SIZE / T);
  //   MatmulKernel_02<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  // }
  // else {
    size_t BM = (M + KERNEL1_BLOCK_SIZE - 1) / KERNEL1_BLOCK_SIZE;
    size_t BP = (P + KERNEL1_BLOCK_SIZE - 1) / KERNEL1_BLOCK_SIZE;
    dim3 grid = dim3(BP, BM, 1);
    dim3 block = dim3(KERNEL1_BLOCK_SIZE, KERNEL1_BLOCK_SIZE, 1);
    MatmulKernel_01<<<grid, block>>>(a.ptr, b.ptr, out->ptr, M, N, P);
  // }

  hipDeviceSynchronize();
  /// END SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////
__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t* out, size_t reduce_size, size_t size){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size){
    size_t start = gid * reduce_size;
    scalar_t tmp = a[start];
    for(size_t i = 1; i < reduce_size; ++i){
      tmp = max(tmp, a[start + i]);
    }
    out[gid] = tmp;
  }
}

void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceMaxKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size, out->size);
  /// END SOLUTION
}

__global__ void ReduceSumKernel(const scalar_t* a, scalar_t* out, size_t reduce_size, size_t size){
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if(gid < size){
    size_t start = gid * reduce_size;
    scalar_t tmp = 0;
    for(size_t i = 0; i < reduce_size; ++i){
      tmp += a[start + i];
    }
    out[gid] = tmp;
  }
}


void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ReduceSumKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, reduce_size, out->size);
  /// END SOLUTION
}

}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
